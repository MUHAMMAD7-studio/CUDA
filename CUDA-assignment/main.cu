#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <stdlib.h>
#include <time.h>
#include "timer.h"

#define threadNumberPerBlock 1024
#define mask 0xffffffff

__global__ void greatestNumber(unsigned int x, int *deviceArray, int *deviceMutex, int *deviceMaximum) {


    unsigned int threadIndX = threadIdx.x;
    unsigned int wid = threadIndX % 32;
    unsigned int blockD = blockDim.x;
   	unsigned int index = threadIndX + blockIdx.x * blockD;
    __shared__ int local[threadNumberPerBlock];
    local[threadIndX] = deviceArray[index];
	__syncthreads();
    //performing the wrap reduction
    for (int oSet = 32; oSet > 0; oSet /= 2) {
        int value = __shfl_down_sync(mask, local[threadIndX], oSet);
        if(value > local[wid]) {
            local[wid] = value;
        }
        __syncthreads();
    }
    //findling maximum use atomicMax function.
    if(threadIndX == 0){
        atomicMax(deviceMaximum, local[0]);
	}

}


int main(){
    int *hostArray,*hostMaximum,*deviceArray,*deviceMaximum, *deviceMutex;
    unsigned int x = 402653184;
    unsigned int i = 0;
    dim3 blockNumber = x/threadNumberPerBlock;
    dim3 threadNumber = threadNumberPerBlock;
    hostArray=(int*)malloc(x*sizeof(int));
    hipMalloc((void**)&deviceArray,x*sizeof(int));
    hostMaximum=(int*)malloc(sizeof(int));
    hipMalloc((void**)&deviceMaximum,sizeof(int));
    hipMemset(deviceMaximum, 0, sizeof(int));
	hipMalloc((void**)&deviceMutex, sizeof(int));
	hipMemset(deviceMutex, 0, sizeof(int));

   
    //putting numbers in the host array.
    while(i<x){
        hostArray[i]=i;
        i++; 
    }

    // serial code starts
    timespec serialStart , serialFinish , serialTimeSpent;
    clock_gettime( CLOCK_REALTIME , &serialStart);
    unsigned int j=0;

    while(j<x){
			if(hostArray[j] > *hostMaximum) 
                *hostMaximum = hostArray[j];
        j++;
	}
    
    clock_gettime( CLOCK_REALTIME , &serialFinish );
    serialTimeSpent = time_diff(serialStart , serialFinish);
    printf("Maximum number found by serial version is %d \n",*hostMaximum);
    printf("serial time spent %ld.%09ld sec.\n" , serialTimeSpent.tv_sec , serialTimeSpent.tv_nsec);
    //serial code ends



   //parallel code starts
    timespec parallelStart , parallelFinish , parallelTimeSpent;


    timespec parallelCopyStart, parallelCopyFinish, parallelCopyTimeSpent;
    clock_gettime( CLOCK_REALTIME , &parallelCopyStart);

    //copying the host data to device data.
    hipMemcpy(deviceArray,hostArray,x*sizeof(int),hipMemcpyHostToDevice);
    

    clock_gettime( CLOCK_REALTIME , &parallelCopyFinish );
    clock_gettime( CLOCK_REALTIME , &parallelStart);
    greatestNumber<<<blockNumber,threadNumber>>>(x, deviceArray , deviceMutex , deviceMaximum);
    clock_gettime( CLOCK_REALTIME , &parallelFinish );

    //copying the device data to host data.
    hipMemcpy(hostMaximum,deviceMaximum,sizeof(int),hipMemcpyDeviceToHost);
    parallelTimeSpent = time_diff(parallelStart , parallelFinish);
    parallelCopyTimeSpent = time_diff(parallelCopyStart , parallelCopyFinish);

    printf("Maximum number found by parallel version is %d \n",*hostMaximum);
    printf("parallel time spent  %ld.%09ld sec.\n" , parallelTimeSpent.tv_sec , parallelTimeSpent.tv_nsec);
    printf("parallel copy time spent  %ld.%09ld sec.\n" , parallelCopyTimeSpent.tv_sec , parallelCopyTimeSpent.tv_nsec);


    

    //now at the end make the memory free.
    free(hostArray);
    hipFree(deviceArray);     
    free(hostMaximum);
    hipFree(deviceMaximum);     


}
